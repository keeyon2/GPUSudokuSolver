#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <hip/hip_runtime.h>

int main (int arg, char* argv[]) {
    
	int device;
	hipGetDevice(&device);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,device);

    printf("Multi Processor Count: %d", prop.multiProcessorCount);
}
